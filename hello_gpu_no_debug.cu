#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void hello_from_gpu() {
    printf("Hello from GPU thread %d in block %d!\n",
           threadIdx.x, blockIdx.x);
}

int main() {
    printf("Launching kernel WITHOUT error check or sync:\n");
    hello_from_gpu<<<2,4>>>();
    printf("CPU finished!\n");
    return 0;
}
