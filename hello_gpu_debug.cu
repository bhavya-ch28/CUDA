#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void hello_from_gpu() {
    printf("Hello from GPU thread %d in block %d!\n",
           threadIdx.x, blockIdx.x);
}

int main() {
    printf("Launching correct kernel:\n");
    hello_from_gpu<<<2,4>>>();
    hipError_t err = hipGetLastError();
    if (err != hipSuccess)
        printf("Error: %s\n", hipGetErrorString(err));
    hipDeviceSynchronize();

    printf("All done from CPU!\n");
    return 0;
}
